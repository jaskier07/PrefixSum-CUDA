#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>

hipError_t sumPrefix(unsigned long long* input, unsigned long long* results, long vector_size, int block_size);

bool checkForError(const hipError_t cudaStatus, const char text[]) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		return true;
	}
	return false;
}

__global__ 
void prefix_sum_kernel(unsigned long long* input, unsigned long long* results, unsigned long long exp, int vector_size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < vector_size) {
		if (i < exp) {
			results[i] = input[i];
		}
		else {
			results[i] = input[i] + input[i - exp];
		}
	}
}

__global__
void copy_kernel(unsigned long long* input, unsigned long long* results, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) {
		input[i] = results[i];
	}
}

bool test(unsigned long long* results, unsigned long long* input, long arraySize) {
	bool prefix_sum_ok = true;
	unsigned long long sum = 1;
	printf("\n\n");

	for (int i = 1; i < arraySize; i++) {
		sum += input[i];
		if (!(results[i] == sum)) {
			printf("BLAD! NIE ZGADZA SIE! oczekiwana = %lld, dostalem = %lld\n", sum, results[i]);
			prefix_sum_ok = false;
		}
	}
	return prefix_sum_ok;
}

int main()
{
	long vector_size;
	long block_size;

	printf("Podaj rozmiar wektora >>> ");
	scanf("%d", &vector_size);
	printf("Podaj liczbe watkow w bloku >>> ");
	scanf("%d", &block_size);

	//const long vector_size = 67108864+2; // 8 digits, max 99999999
	//const long block_size = 512;
	
	unsigned long long* input = (unsigned long long*)malloc(vector_size * sizeof(unsigned long long));
	unsigned long long* results = (unsigned long long*)malloc(vector_size * sizeof(unsigned long long));
	
	for (int i = 0; i < vector_size; i++) {
		input[i] = i + 1;
	}
	
	hipError_t cudaStatus = sumPrefix(input, results, vector_size, block_size);
	if (checkForError(cudaStatus, "sumPrefix failed!")) {
		return 1;
	}

	test(results, input, vector_size);
	printf("%lld, ", results[vector_size - 1]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (checkForError(cudaStatus, "hipDeviceReset failed!")) {
		return 1;
	}

	return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t sumPrefix(unsigned long long* input, unsigned long long* results, long vector_size, int block_size)
{
	int num_blocks = (vector_size + block_size - 1) / block_size;
	int iterations = ceil((int)log2(vector_size));
	unsigned long long exp = 1;
	int i;
	 
	printf("iterations = %d, vector_size = %d, block_size = %d, num_blocks = %d", iterations, vector_size, block_size, num_blocks);

	hipError_t cudaStatus;
	unsigned long long* dev_input = 0;
	unsigned long long* dev_results = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (checkForError(cudaStatus, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?")) {
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_input, vector_size * sizeof(unsigned long long));
	if (checkForError(cudaStatus, "hipMalloc (dev_input) failed!")) {
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_results, vector_size * sizeof(unsigned long long));
	if (checkForError(cudaStatus, "hipMalloc (dev_results) failed!")) {
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, input, vector_size * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if (checkForError(cudaStatus, "hipMemcpy (host -> dev, dev_input) failed!")) {
		goto Error;
	}
	
	printf("\n\nSTART");
	for (i = 0; i <= iterations; i++) {
		prefix_sum_kernel << <num_blocks, block_size >> > (dev_input, dev_results, exp, vector_size);
		exp *= 2;

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus, "prefix_sum_kernel launch failed!")) {
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus, "hipDeviceSynchronize on \"prefix_sum_kernel\" returned error code.")) {
			goto Error;
		}

		copy_kernel << <num_blocks, block_size >> > (dev_input, dev_results, vector_size);
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus,  "copy_kernel launch failed")) {
			goto Error;
		}

		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus,  "hipDeviceSynchronize on \"copy_kernel\" returned error code")) {
			goto Error;
		}
	}
	printf("\nSTOP");

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, dev_results, vector_size * sizeof(unsigned long long), hipMemcpyDeviceToHost);
	if (checkForError(cudaStatus, "hipMemcpy (dev -> host, dev_results) failed!")) {
		goto Error;
	}

Error:
	hipFree(dev_input);
	hipFree(dev_results);

	return cudaStatus;
}
