#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>

hipError_t sumPrefix(unsigned long* input, unsigned long* results, long vector_size, int block_size);

bool checkForError(const hipError_t cudaStatus, const char text[], unsigned long* dev_input, unsigned long* dev_results) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		hipFree(dev_input);
		hipFree(dev_results);
		return true;
	}
	return false;
}

bool checkForError(const hipError_t cudaStatus, const char text[]) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		return true;
	}
	return false;
}

__global__ 
void prefix_sum_kernel(unsigned long* input, unsigned long* results, unsigned long exp, int vector_size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < vector_size) {
		if (i < exp) {
			results[i] = input[i];
		}
		else {
			results[i] = input[i] + input[i - exp];
		}
	}
}

__global__
void copy_kernel(unsigned long* input, unsigned long* results, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) {
		input[i] = results[i];
	}
}

bool test(unsigned long* results, unsigned long* input, long arraySize) {
	bool prefix_sum_ok = true;
	unsigned long sum = 0;
	printf("\n\n");

	for (int i = 0; i < arraySize; i++) {
		sum += input[i];
		if (!(results[i] == sum)) {
			printf("BLAD! NIE ZGADZA SIE! oczekiwana = %ld, dostalem = %ld\n", sum, results[i]);
			prefix_sum_ok = false;
		}
	}
	return prefix_sum_ok;
}

int main()
{
	long vector_size; // 111111111
	long block_size; // 512

	printf("Podaj rozmiar wektora >>> ");
	scanf("%d", &vector_size);
	printf("Podaj liczbe watkow w bloku >>> ");
	scanf("%d", &block_size);
	
	unsigned long* input = (unsigned long*)malloc(vector_size * sizeof(unsigned long));
	unsigned long* results = (unsigned long*)malloc(vector_size * sizeof(unsigned long));
	
	for (int i = 0; i < vector_size; i++) {
		input[i] = i + 99999;//i + 1;
	}
	
	hipError_t cudaStatus = sumPrefix(input, results, vector_size, block_size);
	if (checkForError(cudaStatus, "sumPrefix failed!")) {
		return 1;
	}

	test(results, input, vector_size);
	printf("%ld, ", results[vector_size - 1]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (checkForError(cudaStatus, "hipDeviceReset failed!")) {
		return 1;
	}

	return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t sumPrefix(unsigned long* input, unsigned long* results, long vector_size, int block_size)
{
	int num_blocks = (vector_size + block_size - 1) / block_size;
	int iterations = ceil((int)log2((float)vector_size));
	unsigned long exp = 1;
	int i;
	 
	printf("iterations = %d, vector_size = %d, block_size = %d, num_blocks = %d", iterations, vector_size, block_size, num_blocks);

	hipError_t cudaStatus;
	unsigned long* dev_input = 0;
	unsigned long* dev_results = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (checkForError(cudaStatus, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?", dev_input, dev_results)) {
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_input, vector_size * sizeof(unsigned long));
	if (checkForError(cudaStatus, "hipMalloc (dev_input) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_results, vector_size * sizeof(unsigned long));
	if (checkForError(cudaStatus, "hipMalloc (dev_results) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, input, vector_size * sizeof(unsigned long), hipMemcpyHostToDevice);
	if (checkForError(cudaStatus, "hipMemcpy (host -> dev, dev_input) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}
	
	printf("\n\nSTART"); fflush(stdout);
	for (i = 0; i <= iterations; i++) {
		prefix_sum_kernel << <num_blocks, block_size >> > (dev_input, dev_results, exp, vector_size);
		exp *= 2;

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus, "prefix_sum_kernel launch failed!", dev_input, dev_results)) {
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus, "hipDeviceSynchronize on \"prefix_sum_kernel\" returned error code.", dev_input, dev_results)) {
			return cudaStatus;
		}

		copy_kernel << <num_blocks, block_size >> > (dev_input, dev_results, vector_size);
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus, "copy_kernel launch failed", dev_input, dev_results)) {
			return cudaStatus;
		}

		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus, "hipDeviceSynchronize on \"copy_kernel\" returned error code", dev_input, dev_results)) {
			return cudaStatus;
		}
	}
	printf("\nSTOP"); fflush(stdout);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, dev_results, vector_size * sizeof(unsigned long), hipMemcpyDeviceToHost);
	if (checkForError(cudaStatus, "hipMemcpy (dev -> host, dev_results) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	hipFree(dev_input);
	hipFree(dev_results);

	return cudaStatus;
}
