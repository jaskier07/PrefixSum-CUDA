#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>

#define CUSTOM_DATA false

hipError_t sumPrefix(unsigned long* input, unsigned long* results, long vector_size, int block_size);

bool checkForError(const hipError_t cudaStatus, const char text[], unsigned long* dev_input, unsigned long* dev_results) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		hipFree(dev_input);
		hipFree(dev_results);
		return true;
	}
	return false;
}

bool checkForError(const hipError_t cudaStatus, const char text[]) {
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n\n%s \nError code: %d \nStatus: %s \n\n", text, cudaStatus, hipGetErrorString(cudaStatus));
		return true;
	}
	return false;
}

__global__
void prefix_sum_kernel(unsigned long* input, unsigned long* results, unsigned long exp, int vector_size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < vector_size) {
		if (i < exp) {
			results[i] = input[i];
		}
		else {
			results[i] = input[i] + input[i - exp];
		}
	}
}

__global__
void copy_kernel(unsigned long* input, unsigned long* results, int size) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < size) {
		input[i] = results[i];
	}
}

__global__ 
void scan_simple(unsigned long *g_odata, unsigned long *g_idata, int n)
{
	extern __shared__ unsigned long temp[];  // allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;

	temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
	temp[2 * thid + 1] = g_idata[2 * thid + 1];

	// n/2 bo sumujemy naraz dwa elementy, d > 0 bo ko�czymy sumowanie w korzeniu, d/=2 bo 
	for (int d = n /2; d >= 0; d /= 2)  // build sum in place up the tree; d = depth, idziemy od li�ci do korzenia
	{
		__syncthreads();
		if (thid < d)
		{
			// bierzemy dwa elementy i sumujemy
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	if (thid == 0) { 
		temp[n - 1] = 0; 
	} // clear the last element

	for (int d = 1; d <= n; d *= 2) // traverse down tree & build scan
	{
		offset /= 2;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			unsigned long t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
	g_odata[2 * thid + 1] = temp[2 * thid + 1];
}

bool test(unsigned long* results, unsigned long* input, long arraySize) {
	bool prefix_sum_ok = true;
	unsigned long sum = 0;
	printf("\n\n");

	for (int i = 0; i < arraySize; i++) {
		sum += input[i];
		if (!(results[i] == sum)) {
			printf("BLAD! NIE ZGADZA SIE! oczekiwana = %ld, dostalem = %ld\n", sum, results[i]);
			prefix_sum_ok = false;
		}
	}
	return prefix_sum_ok;
}

int main()
{
	long vector_size; // 111111111
	long block_size; // 512

	if (CUSTOM_DATA) {
		printf("Podaj rozmiar wektora >>> ");
		scanf("%d", &vector_size);
		printf("Podaj liczbe watkow w bloku >>> ");
		scanf("%d", &block_size);
	}
	else {
		vector_size = 8;
		block_size = 8;//4;
	}

	unsigned long* input = (unsigned long*)malloc(vector_size * sizeof(unsigned long));
	unsigned long* results = (unsigned long*)malloc(vector_size * sizeof(unsigned long));

	for (int i = 0; i < vector_size; i++) {
		if (CUSTOM_DATA) {
			input[i] = i + 99999; //i + 1;
		}
		else {
			input[i] = i + 1;
		}
	}

	hipError_t cudaStatus = sumPrefix(input, results, vector_size, block_size);
	if (checkForError(cudaStatus, "sumPrefix failed!")) {
		return 1;
	}

	test(results, input, vector_size);
	printf("%ld, ", results[vector_size - 1]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (checkForError(cudaStatus, "hipDeviceReset failed!")) {
		return 1;
	}

	return 0;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t sumPrefix(unsigned long* input, unsigned long* results, long vector_size, int block_size)
{
	int num_blocks = (vector_size + block_size - 1) / block_size;
	int iterations = ceil((int)log2((float)vector_size));
	unsigned long exp = 1;
	int i;

	printf("iterations = %d, vector_size = %d, block_size = %d, num_blocks = %d", iterations, vector_size, block_size, num_blocks);

	hipError_t cudaStatus;
	unsigned long* dev_input = 0;
	unsigned long* dev_results = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (checkForError(cudaStatus, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?", dev_input, dev_results)) {
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_input, vector_size * sizeof(unsigned long));
	if (checkForError(cudaStatus, "hipMalloc (dev_input) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_results, vector_size * sizeof(unsigned long));
	if (checkForError(cudaStatus, "hipMalloc (dev_results) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_input, input, vector_size * sizeof(unsigned long), hipMemcpyHostToDevice);
	if (checkForError(cudaStatus, "hipMemcpy (host -> dev, dev_input) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	printf("\n\nSTART"); fflush(stdout);
	scan_simple << <num_blocks, block_size, vector_size >> > (dev_results, dev_input, vector_size);
	/*
	for (i = 0; i <= iterations; i++) {
		prefix_sum_kernel << <num_blocks, block_size >> > (dev_input, dev_results, exp, vector_size);
		exp *= 2;

		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus, "prefix_sum_kernel launch failed!", dev_input, dev_results)) {
			return cudaStatus;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus, "hipDeviceSynchronize on \"prefix_sum_kernel\" returned error code.", dev_input, dev_results)) {
			return cudaStatus;
		}

		copy_kernel << <num_blocks, block_size >> > (dev_input, dev_results, vector_size);
		cudaStatus = hipGetLastError();
		if (checkForError(cudaStatus, "copy_kernel launch failed", dev_input, dev_results)) {
			return cudaStatus;
		}

		cudaStatus = hipDeviceSynchronize();
		if (checkForError(cudaStatus, "hipDeviceSynchronize on \"copy_kernel\" returned error code", dev_input, dev_results)) {
			return cudaStatus;
		}
	}
	*/
	printf("\nSTOP"); fflush(stdout);

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, dev_results, vector_size * sizeof(unsigned long), hipMemcpyDeviceToHost);
	if (checkForError(cudaStatus, "hipMemcpy (dev -> host, dev_results) failed!", dev_input, dev_results)) {
		return cudaStatus;
	}

	hipFree(dev_input);
	hipFree(dev_results);

	return cudaStatus;
}
